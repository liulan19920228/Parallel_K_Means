#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "kmeans.h"


__global__ static
void find_nearest_cluster(int dimension,
                          int numObjs,
                          int numClusters,
                          float *data,
                          float *DEVICEcenter,
                          int *newmembership)
{
    
    int datatId = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (objectId < numObjs) {
        int   i;
        float distance, min_dist = 0.0;
        
        for(i=0; i<dimension; i++)
        {
            min_dist +=(data[numObjs * i + dataId] - DEVICEcenter[numClusters * i]) *
            (data[numObjs * i + dataId] - DEVICEcenter[numClusters * i]);
        }
            
        for (j=1; j<numClusters; j++) {
            distance = 0.0;
            for(i=0; i<dimension; i++)
            {
                distance +=(data[numObjs * i + dataId] - DEVICEcenter[numClusters * i + j]) *
                (data[numObjs * i + dataId] - DEVICEcenter[numClusters * i + j]);
            }
            if (distance < min_dist) {
                min_dist = distance;
                newmembership[dataId] = i;
            }
        }
    
         __syncthreads();
    }
}

float** cuda_kmeans(float **data, int dimension, int numObjs, int numClusters, float threshold                 , int *membership, int *num_iterations)
{
    int      i, j, num_iterations=0;
    int     *clustersize;
    float    delta, **center, **clustersum;
    float  **datatranspose;
    float  **centertranspose;
    float *DEVICEdata;
    float *DEVICEcenter;
    int *DEVICEmembership;
    int *newmembership

    center    = (float**) malloc(numClusters *sizeof(float*));
    malloc2D(datatranspose, dimension, numObjs, float);
    for (i = 0; i < dimension; i++) {
        for (j = 0; j < numObjs; j++) {
            datatranspose[i][j] = data[j][i];
        }
    }
    
    malloc2D(centertranspose, dimension, numClusters, float);
    for (i = 0; i < dimension; i++) {
        for (j = 0; j < numClusters; j++) {
            centertranspose[i][j] = datatranspose[i][j];
        }
    }
    
    clustersize = (int*) calloc(numClusters, sizeof(int));
    calloc2D(clustersum, numClusters, dimension);
    newmembership = (int*) calloc(numObjs, sizeof(int));
    
    const unsigned int numThreadsPerClusterBlock = 128;
    const unsigned int numClusterBlocks =(numObjs + numThreadsPerClusterBlock - 1) /numThreadsPerClusterBlock;

    hipMalloc(&DEVICEdata, numObjs*dimension*sizeof(float));
    hipMalloc(&DEVICEcenter, numClusters*dimension*sizeof(float));
    hipMalloc(&DEVICEmembership, numObjs*sizeof(int));
    hipMemcpy(DEVICEdata, datatranspose[0],
                         numObjs*dimension*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(DEVICEmembership, membership,
                         numObjs*sizeof(int), hipMemcpyHostToDevice);
    
    do {
        hipMemcpy(DEVICEcenter, centertranspose[0],
                             numClusters*dimension*sizeof(float), hipMemcpyHostToDevice);
        
        find_nearest_cluster
        <<< numClusterBlocks, numThreadsPerClusterBlock>>>
        (dimension, numObjs, numClusters,
         DEVICEdata, DEVICEcenter, DEVICEmembership);
        
        hipDeviceSynchronize();
        checkLastCudaError();
        
        delta=0.0;
        
        hipMemcpy(newmembership, DEVICEmembership,
                   numObjs*sizeof(int), hipMemcpyDeviceToHost)
        
        for(i=0; i<numObjs; i++)
        {
            if(numiteration == 0){
                delta =float(numObjs);
                membership[i]=newmembership[i];
                clustersize[membership[i]]++;
                clustersum[membership[i]] += data[i];
            }
            
            else if(membership[i] != newmembership[i]){
                delta += 1.0;
                clustersize[newmembership[i]] ++;
                clustersize[membership[i]] --;
                for(j=0; j<dimension; j++){
                    clustersum[newmembership][j] -= data[i][j];
                    clustersum[membership[i]][j] -= data[i][j];
                }
                membership[i] = newmembership[i];
            }
        }
        
        
        /* average the sum to compute new cluster centers*/
        for (i=0; i<numClusters; i++) {
            for (j=0; j<dimension; j++) {
                if (clustersize[i] > 0)
                    centertranspose[j][i] = clustersum[i][j] / clustersize[i];
            }
        }        
        delta /= numObjs;
    } while (delta > threshold && numiterations++ < 500);//Max number of iteration
    
    *num_iterations = numiterations + 1;
    
    for (i = 0; i < numClusters; i++) {
        for (j = 0; j < dimension; j++) {
            clusters[i][j] = clustertranspose[j][i];
        }
    }
    
    hipFree(DEVICEdata);
    hipFree(DEVICEcenter);
    hipFree(DEVICEmembership);
    
    free(datatranspose[0]);
    free(datatranspose);
    free(centertranspose[0]);
    free(centertranspose);
    free(clustersum[0]);
    free(clustersum);
    free(clustersize);
    free(newmembership)
    return center;
}
